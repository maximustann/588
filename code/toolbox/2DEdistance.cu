#include <hip/hip_runtime.h>
#include <math.h>
#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#define SIZE 1024

float *readFiletoArray(char *filename, int *size)
{
	FILE *fstream = fopen(filename, "r");
	int i = 0;
	char buf[SIZE];
	char *record;
	char *numbers, *number1, *number2;
	float *mat;
	if(fstream == NULL)
	{
		perror("open failed:");
		return NULL;
	}
	mat = (float *)malloc(SIZE * sizeof(float));
	memset(mat, 0, SIZE * sizeof(float));
	while((record = fgets(buf, sizeof(buf), fstream))!=NULL)
	{
		numbers = strtok(record, "\n");
		number1 = strtok(numbers, ",");
		number2 = strtok(NULL, " ");
		mat[i] = atof(number1);
		++i;
		mat[i] = atof(number2);
		++i;
	}
	//for(int iter = 0; iter < i; iter++)
	//{
		//printf("%G\n", mat[iter]);
	//}
	*size = i;
	fclose(fstream);
	return mat;
}

__global__ void step1(float *d_A, size_t pitch, int rows, int cols, float mat2row, float mat2col){
	int r = blockIdx.y * blockDim.y + threadIdx.y;
	int c = blockIdx.x * blockDim.x + threadIdx.x;
	if((r < rows) && (c < cols)){
		float *Row = (float *)((char *)d_A + r*pitch);
		float elem = Row[c];
		if(c == 0){
			d_A[r * 2 + c] = (elem - mat2row) * (elem - mat2row);
		}else{
			d_A[r * 2 + c] = (elem - mat2col) * (elem - mat2col);
		}
	}
}

__global__ void step2(float *d_A, float *result, size_t pitch, int rows, int cols){
	int r = threadIdx.x;
	result[r] = d_A[r*2] + d_A[r*2 + 1];
	//printf("result[%d] = %G\n", r, result[r]);
}

float min(float *arr, size_t length) {
    // returns the minimum value of array
    size_t i;
    float minimum = arr[0];
    for (i = 1; i < length; ++i) {
        if (minimum > arr[i]) {
            minimum = arr[i];
        }
    }
    return minimum;
}

float igd(char *refFront, char *front)
{
	float *mat1;
	float *mat2;
	float *d_mat1,*d_mat2;
	float *Curesult, *result;
	int *mat1size, *mat2size;
	int columnCount = 2, rowCount;
	mat1size = (int *)malloc(sizeof(int));
	*mat1size = 0;
	mat2size = (int *)malloc(sizeof(int));
	*mat2size = 0;
	float igd_value;

	
	mat1 = readFiletoArray(refFront, mat1size);
	mat2 = readFiletoArray(front, mat2size);
	size_t pitch;
	hipMallocPitch((void **)&d_mat2, &pitch, sizeof(int)*columnCount, *mat2size);
	hipMalloc((void **)&Curesult, *mat2size * sizeof(float) / 2);
	result = (float *)malloc(*mat2size * sizeof(float) / 2);
	memset(result, 0, *mat2size * sizeof(float) / 2);
	dim3 block(2,*mat2size/2);
	dim3 grid(1, 1);

	for(int i = 0; i < *mat1size;i += 2)
	{
		//copy front to cuda mem
		hipMemcpy2D(d_mat2, pitch, mat2, sizeof(float)*columnCount, sizeof(float)*columnCount, *mat2size, hipMemcpyHostToDevice);
		step1<<<grid, block>>>(d_mat2, pitch, *mat2size, columnCount, mat1[i], mat1[i + 1]);
		step2<<<1, *mat2size/2>>>(d_mat2, Curesult, pitch, *mat2size, columnCount);
		hipMemcpy(result, Curesult, *mat2size * sizeof(float) / 2, hipMemcpyDeviceToHost);
		hipError_t error = hipGetLastError();
		if(error != hipSuccess){
			printf("%s\n", hipGetErrorString(error));
		}
		//printf("igd_value = %G\n", sqrt(min(result, *mat2size/2)));
		igd_value += sqrt(min(result, *mat2size/2));
	}
	//printf("igd_value = %G\n", igd_value);
	igd_value = igd_value / (*mat1size /2 );
	hipDeviceSynchronize();

	hipFree(d_mat1);
	hipFree(d_mat2);
	hipFree(Curesult);
	free(result);
	free(mat1);
	free(mat2);
	free(mat1size);
	free(mat2size);
	return igd_value;
}

char* itoa(int i, char b[]){
    char const digit[] = "0123456789";
    char* p = b;
    if(i<0){
        *p++ = '-';
        i *= -1;
    }
    int shifter = i;
    do{ //Move to where representation ends
        ++p;
        shifter = shifter/10;
    }while(shifter);
    *p = '\0';
    do{ //Move back, inserting digits as u go
        *--p = digit[i%10];
        i = i/10;
    }while(i);
    return b;
}
float mean(float *a, int num)
{
	float value= 0;
	for(int i = 0; i < num; i++)
	{
		value += a[i];
	}
	return(value/num);
}

int create_csv(char *algorithm, char *problem, float *array, int n)
{

	FILE *fp;

	int i;

	char filename[SIZE] = "/home/st-james1/tanboxi/588_project/code/";
	strcat(filename, algorithm);
	strcat(filename, "/result/");
	strcat(filename, problem);
	strcat(filename, "/best/igd.csv");

	fp = fopen(filename, "w+");
	if(fp == NULL)
	{
		perror("open failed: ");
		return -1;
	}

	for(i = 0;i < n; i++){
		fprintf(fp,"%f,%d\n", array[i], i+1);
	}

	printf("finished!\n");
	fclose(fp);
	return 0;
}


int main(int argc, char *argv[])
{

	float igd_value[SIZE];
	char filename[SIZE];
	char base[SIZE] = "/home/st-james1/tanboxi/588_project/code/";
	char filedir[SIZE];
	char refFrontfilename[SIZE];
	int generation = 50;
	int run = 40;
	float runigd[run];
	float genigd[generation];
	char num[4];
	//float igd_value;
	//igd_value = igd("2.csv", "front.csv");
	//printf("igd = %G\n", igd_value);
	memcpy(refFrontfilename, base, strlen(base));
	strcat(refFrontfilename, "dataset/trueFront/");
	strcat(refFrontfilename, argv[2]);
	strcat(refFrontfilename, ".csv");
	for(int j = 1; j <= generation; j++)
	{
		for(int i = 1; i <= run; i++)
		{
			memcpy(filedir, base, strlen(base));
			strcat(filedir, argv[1]); //algorithm name
			strcat(filedir, "/logData/");
			memcpy(filename, filedir, strlen(filedir));
			strcat(filename, argv[2]); //problem N
			strcat(filename, "/");
			strcat(filename, itoa(i, num));
			strcat(filename, "/");
			strcat(filename, itoa(j, num));
			strcat(filename, "/front.csv");
			//printf("%s\n", filename);
			runigd[i-1] = igd(refFrontfilename, filename);
			//printf("runigd = %G\n", runigd[i-1]);
			memset(filedir, 0, strlen(filedir));
			memset(filename, 0, strlen(filename));
		}
		genigd[j-1] = mean(runigd, run);
	}
	//for(int j = 0; j < generation; j++)
	//{
//		printf("igd = %G, j = %d\n", genigd[j], j);
//	}

	create_csv(argv[1], argv[2], genigd, generation);
	return 0;
}
